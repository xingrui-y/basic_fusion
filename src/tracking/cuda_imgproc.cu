#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
#include "tracking/cuda_imgproc.h"
#include "utils/safe_call.h"
#include "math/matrix_type.h"
#include "math/vector_type.h"
#include "data_struct/intrinsic_matrix.h"

namespace fusion
{

FUSION_HOST inline dim3 createGrid(dim3 block, int cols, int rows)
{
    return dim3(div_up(cols, block.x), div_up(rows, block.y));
}

FUSION_DEVICE inline Vector4c renderPoint(
    const Vector3f &point,
    const Vector3f &normal,
    const Vector3f &image,
    const Vector3f &light_pos)
{
    Vector3f colour(4.f / 255.f, 2.f / 255.f, 2.f / 255.f);
    if (!isnan(point.x))
    {
        const float Ka = 0.3f; //ambient coeff
        const float Kd = 0.5f; //diffuse coeff
        const float Ks = 0.2f; //specular coeff
        const float n = 20.f;  //specular power

        const float Ax = image.x; //ambient color,  can be RGB
        const float Dx = image.y; //diffuse color,  can be RGB
        const float Sx = image.z; //specular color, can be RGB
        const float Lx = 1.f;     //light color

        Vector3f L = normalised(light_pos - point);
        Vector3f V = normalised(Vector3f(0.f, 0.f, 0.f) - point);
        Vector3f R = normalised(2 * normal * (normal * L) - L);

        float Ix = Ax * Ka * Dx + Lx * Kd * Dx * fmax(0.f, (normal * L)) + Lx * Ks * Sx * pow(fmax(0.f, (R * V)), n);
        colour = Vector3f(Ix, Ix, Ix);
    }

    return Vector4c(
        static_cast<unsigned char>(__saturatef(colour.x) * 255.f),
        static_cast<unsigned char>(__saturatef(colour.y) * 255.f),
        static_cast<unsigned char>(__saturatef(colour.z) * 255.f),
        255);
}

FUSION_KERNEL void renderSceneK(
    const cv::cuda::PtrStep<Vector4f> vmap,
    const cv::cuda::PtrStep<Vector4f> nmap,
    const Vector3f light_pos,
    cv::cuda::PtrStepSz<Vector4c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= dst.cols || y >= dst.rows)
        return;

    Vector3f point = ToVector3(vmap.ptr(y)[x]);
    Vector3f normal = ToVector3(nmap.ptr(y)[x]);
    Vector3f pixel(1.f);

    dst.ptr(y)[x] = renderPoint(point, normal, pixel, light_pos);
}

void renderScene(const cv::cuda::GpuMat vmap, const cv::cuda::GpuMat nmap, cv::cuda::GpuMat &image)
{
    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    if (image.empty())
        image.create(vmap.rows, vmap.cols, CV_8UC4);

    renderSceneK<<<grid, block>>>(vmap, nmap, Vector3f(5, 5, 5), image);
}

FUSION_KERNEL void renderSceneTexturedK(
    const cv::cuda::PtrStep<Vector4f> vmap,
    const cv::cuda::PtrStep<Vector4f> nmap,
    const cv::cuda::PtrStep<Vector3c> image,
    const Vector3f light_pos,
    cv::cuda::PtrStepSz<Vector4c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= dst.cols || y >= dst.rows)
        return;

    Vector3f point = ToVector3(vmap.ptr(y)[x]);
    Vector3f normal = ToVector3(nmap.ptr(y)[x]);
    Vector3f pixel = ToVector3f(image.ptr(y)[x]) / 255.f;

    dst.ptr(y)[x] = renderPoint(point, normal, pixel, light_pos);
}

void renderSceneTextured(const cv::cuda::GpuMat vmap, const cv::cuda::GpuMat nmap, const cv::cuda::GpuMat image, cv::cuda::GpuMat &out)
{
    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    if (out.empty())
        out.create(vmap.rows, vmap.cols, CV_8UC4);

    renderSceneTexturedK<<<grid, block>>>(vmap, nmap, image, Vector3f(5, 5, 5), out);
}

FUSION_KERNEL void ToSemiDenseImageK(
    const cv::cuda::PtrStepSz<float> image,
    const cv::cuda::PtrStepSz<float> intensity_dx,
    const cv::cuda::PtrStepSz<float> intensity_dy,
    cv::cuda::PtrStepSz<float> semi,
    float th_dx, float th_dy)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= image.cols || y >= image.rows)
        return;

    semi.ptr(y)[x] = 255;

    auto dx = intensity_dx.ptr(y)[x];
    auto dy = intensity_dy.ptr(y)[x];

    if (dx > th_dx || dy > th_dy)
    {
        semi.ptr(y)[x] = image.ptr(y)[x];
    }
}

void convert_image_to_semi_dense(const cv::cuda::GpuMat image, const cv::cuda::GpuMat dx, const cv::cuda::GpuMat dy, cv::cuda::GpuMat &semi, float th_dx, float th_dy)
{
    if (semi.empty())
        semi.create(image.size(), image.type());

    dim3 block(8, 4);
    dim3 grid = createGrid(block, image.cols, image.rows);

    ToSemiDenseImageK<<<grid, block>>>(image, dx, dy, semi, th_dx, th_dy);
}

void build_semi_dense_pyramid(const std::vector<cv::cuda::GpuMat> image_pyr, const std::vector<cv::cuda::GpuMat> dx_pyr, const std::vector<cv::cuda::GpuMat> dy_pyr, std::vector<cv::cuda::GpuMat> &semi_pyr, float th_dx, float th_dy)
{
    if (semi_pyr.size() != image_pyr.size())
        semi_pyr.resize(image_pyr.size());

    for (int level = 0; level < image_pyr.size(); ++level)
    {
        convert_image_to_semi_dense(image_pyr[level], dx_pyr[level], dy_pyr[level], semi_pyr[level], th_dx, th_dy);
    }
}

FUSION_DEVICE inline Vector3c interpolate_bilinear(const cv::cuda::PtrStepSz<Vector3c> image, float x, float y)
{
    int u = std::floor(x), v = std::floor(y);
    float coeff_x = x - (float)u, coeff_y = y - (float)v;
    Vector3f result = ToVector3f((image.ptr(v)[u] * (1 - coeff_x) + image.ptr(v)[u + 1] * coeff_x) * (1 - coeff_y) +
                                 (image.ptr(v + 1)[u] * (1 - coeff_x) + image.ptr(v + 1)[u + 1] * coeff_x) * coeff_y);
    return ToVector3c(result);
}

FUSION_KERNEL void warp_image_kernel(const cv::cuda::PtrStepSz<Vector3c> src,
                                     const cv::cuda::PtrStep<Vector4f> vmap_dst,
                                     const Matrix3x4f pose,
                                     const IntrinsicMatrix K,
                                     cv::cuda::PtrStep<Vector3c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= src.cols || y >= src.rows)
        return;

    dst.ptr(y)[x] = Vector3c(0);
    Vector3f dst_pt_src = pose(ToVector3(vmap_dst.ptr(y)[x]));

    float u = K.fx * dst_pt_src.x / dst_pt_src.z + K.cx;
    float v = K.fy * dst_pt_src.y / dst_pt_src.z + K.cy;
    if (u >= 1 && v >= 1 && u < src.cols - 1 && v < src.rows - 1)
    {
        dst.ptr(y)[x] = interpolate_bilinear(src, u, v);
    }
}

void warp_image(const cv::cuda::GpuMat src, const cv::cuda::GpuMat vmap_dst, const Sophus::SE3d pose, const IntrinsicMatrix K, cv::cuda::GpuMat &dst)
{
    if (dst.empty())
        dst.create(src.size(), src.type());

    dim3 block(8, 4);
    dim3 grid = createGrid(block, src.cols, src.rows);

    warp_image_kernel<<<grid, block>>>(src, vmap_dst, pose.cast<float>().matrix3x4(), K, dst);
}

FUSION_HOST void filterDepthBilateral(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::bilateralFilter(src, dst, 5, 1, 1);
}

FUSION_HOST void pyrDownDepth(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::resize(src, dst, cv::Size(0, 0), 0.5, 0.5);
}

FUSION_HOST void pyrDownImage(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::pyrDown(src, dst);
}

FUSION_HOST void pyrDownVMap(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::resize(src, dst, cv::Size(0, 0), 0.5, 0.5);
}

FUSION_KERNEL void computeDerivativeK(
    cv::cuda::PtrStepSz<float> image,
    cv::cuda::PtrStep<float> dx,
    cv::cuda::PtrStep<float> dy)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= image.cols - 1 || y >= image.rows - 1)
        return;

    int x10 = max(x - 1, 0);
    int x01 = min(x + 1, image.cols);
    int y10 = max(y - 1, 0);
    int y01 = min(y + 1, image.rows);

    dx.ptr(y)[x] = (image.ptr(y)[x01] - image.ptr(y)[x10]) * 0.5;
    dy.ptr(y)[x] = (image.ptr(y01)[x] - image.ptr(y10)[x]) * 0.5;
}

FUSION_HOST void computeDerivative(const cv::cuda::GpuMat image, cv::cuda::GpuMat &dx, cv::cuda::GpuMat &dy)
{
    if (dx.empty())
        dx.create(image.size(), image.type());
    if (dy.empty())
        dy.create(image.size(), image.type());

    dim3 block(8, 8);
    dim3 grid(div_up(image.cols, block.x), div_up(image.rows, block.y));

    computeDerivativeK<<<grid, block>>>(image, dx, dy);
}

FUSION_KERNEL void backProjectDepthK(const cv::cuda::PtrStepSz<float> depth, cv::cuda::PtrStep<Vector4f> vmap, IntrinsicMatrix intrinsics)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x > depth.cols - 1 || y > depth.rows - 1)
        return;

    vmap.ptr(y)[x] = Vector4f(nanf("NAN"), nanf("NAN"), nanf("NAN"), -1.0f);
    float z = depth.ptr(y)[x];
    // z = (z == z) ? z : nanf("NAN");
    if (z > 0.3f && z < 5.0f)
    {
        vmap.ptr(y)[x] = Vector4f(z * (x - intrinsics.cx) * intrinsics.invfx, z * (y - intrinsics.cy) * intrinsics.invfy, z, 1.0f);
    }
}

FUSION_HOST void backProjectDepth(const cv::cuda::GpuMat depth, cv::cuda::GpuMat &vmap, const IntrinsicMatrix &K)
{
    if (vmap.empty())
        vmap.create(depth.size(), CV_32FC4);

    dim3 block(8, 8);
    dim3 grid = createGrid(block, depth.cols, depth.rows);

    backProjectDepthK<<<grid, block>>>(depth, vmap, K);
}

FUSION_KERNEL void computeNMapK(cv::cuda::PtrStepSz<Vector4f> vmap, cv::cuda::PtrStep<Vector4f> nmap)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= vmap.cols - 1 || y >= vmap.rows - 1)
        return;

    int x10 = max(x - 1, 0);
    int x01 = min(x + 1, vmap.cols);
    int y10 = max(y - 1, 0);
    int y01 = min(y + 1, vmap.rows);

    Vector3f v00 = ToVector3(vmap.ptr(y)[x10]);
    Vector3f v01 = ToVector3(vmap.ptr(y)[x01]);
    Vector3f v10 = ToVector3(vmap.ptr(y10)[x]);
    Vector3f v11 = ToVector3(vmap.ptr(y01)[x]);

    nmap.ptr(y)[x] = Vector4f(normalised((v01 - v00).cross(v11 - v10)), 1.f);
}

FUSION_HOST void computeNMap(const cv::cuda::GpuMat vmap, cv::cuda::GpuMat &nmap)
{
    if (nmap.empty())
        nmap.create(vmap.size(), vmap.type());

    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    computeNMapK<<<grid, block>>>(vmap, nmap);
}

__global__ void select_point_with_gradient_kernel(
    const cv::cuda::PtrStepSz<float> intensity,
    const cv::cuda::PtrStep<float> depth,
    const cv::cuda::PtrStep<float> dx,
    const cv::cuda::PtrStep<float> dy,
    cv::cuda::PtrStep<float> mask_out,
    Vector4f *selected_points)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= intensity.cols || y >= intensity.rows)
        return;

    if (sqrt(pow(dx.ptr(y)[x], 2) + pow(dy.ptr(y)[x], 2)) > 2)
    {
        mask_out.ptr(y)[x] = intensity.ptr(y)[x];
    }
    else
    {
        mask_out.ptr(y)[x] = 0;
    }
}

void select_point(
    const cv::cuda::GpuMat intensity,
    const cv::cuda::GpuMat depth,
    const cv::cuda::GpuMat dx,
    const cv::cuda::GpuMat dy,
    Vector4f *selected_points)
{
    const auto cols = intensity.cols;
    const auto rows = intensity.rows;

    dim3 block(8, 8);
    dim3 grid(div_up(cols, block.x), div_up(rows, block.y));

    cv::cuda::GpuMat mask(intensity.size(), intensity.type());

    select_point_with_gradient_kernel<<<grid, block>>>(intensity, depth, dx, dy, mask, selected_points);

    cv::Mat img(mask);
    cv::imshow("img", img);
    cv::waitKey(1);
}

__global__ void check_covisibility_kernel(
    const cv::cuda::PtrStepSz<Vector4f> vmap,
    Matrix3x3f KRKinv, Vector3f Kt,
    uint *num_points)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= vmap.cols || y >= vmap.rows)
        return;

    Vector3f vertex = ToVector3(vmap.ptr(y)[x]);
    Vector3f transformed_point = KRKinv(vertex) + Kt;
    if (transformed_point.x >= 0 &&
        transformed_point.y >= 0 &&
        transformed_point.x < vmap.cols &&
        transformed_point.y < vmap.rows)
    {
        atomicAdd(num_points, 1);
    }
}

float check_covisibility(
    const cv::cuda::GpuMat vmap,
    Eigen::Matrix3f R,
    Eigen::Vector3f t,
    IntrinsicMatrix &K)
{
    const int cols = vmap.cols;
    const int rows = vmap.rows;

    Eigen::Matrix3f Kmat;
    Kmat << K.fx, 0, K.cx,
        0, K.fy, K.cy,
        0, 0, 1;
    Eigen::Matrix3f KR = Kmat * R;
    Eigen::Vector3f Kt = Kmat * t;

    dim3 block(8, 8);
    dim3 grid(div_up(cols, block.x), div_up(rows, block.y));

    uint *num_points;

    hipMalloc((void **)&num_points, sizeof(uint));
    hipMemset(num_points, 0, sizeof(uint));

    check_covisibility_kernel<<<grid, block>>>(vmap, KR, Vector3f(Kt(0), Kt(1), Kt(2)), num_points);

    uint temp = 0;
    hipMemcpy(&temp, num_points, sizeof(uint), hipMemcpyDeviceToHost);

    hipFree(num_points);
    return (float)temp / (cols * rows);
}

__device__ inline float interpolate(const cv::cuda::PtrStep<float> &map, float x, float y)
{
    float u = std::floor(x), v = std::floor(y);
    float coeff_x = x - u, coeff_y = y - v;

    float v00 = map.ptr((int)u)[(int)v];
    float v10 = map.ptr((int)u)[(int)v + 1];
    float v01 = map.ptr((int)u + 1)[(int)v];
    float v11 = map.ptr((int)u + 1)[(int)v + 1];

    return (v00 * (1 - coeff_x) * v10 * coeff_x) * (1 - coeff_y) + (v01 * (1 - coeff_x) + v11 * coeff_x) * coeff_y;
}

__global__ void compute_residual_kernel(
    const cv::cuda::PtrStep<float> ref_image,
    const cv::cuda::PtrStep<Vector4f> ref_vmap,
    const cv::cuda::PtrStep<float> src_image,
    Matrix3x3f KR_ref2src, Vector3f Kt_ref2src,
    cv::cuda::PtrStepSz<float> out_image)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= out_image.cols || y >= out_image.rows)
        return;

    out_image.ptr(y)[x] = 255;

    Vector3f point = ToVector3(ref_vmap.ptr(y)[x]);
    Vector3f project = KR_ref2src(point) + Kt_ref2src;
    if (project.x >= 1 && project.y >= 1 &&
        project.x < out_image.cols - 1 &&
        project.y < out_image.rows - 1)
    {
        float val_interp = interpolate(src_image, project.x, project.y);
        out_image.ptr(y)[x] = abs(val_interp - ref_image.ptr(y)[x]);
    }
}

void compute_residual(
    const cv::cuda::GpuMat ref_image,
    const cv::cuda::GpuMat ref_vmap,
    const cv::cuda::GpuMat src_image,
    const IntrinsicMatrix &K,
    const Eigen::Matrix4d T_ref2src,
    cv::cuda::GpuMat &out_image)
{
    if (out_image.empty())
        out_image.create(ref_image.size(), ref_image.type());

    dim3 block(8, 8);
    dim3 grid(div_up(out_image.cols, block.x), div_up(out_image.rows, block.y));

    Eigen::Matrix3d Kmat;
    Kmat << K.fx, 0, K.cx,
        0, K.fy, K.cy,
        0, 0, 1;

    Eigen::Matrix3d KR = Kmat * T_ref2src.topLeftCorner(3, 3);
    Eigen::Vector3d Kt = Kmat * T_ref2src.topRightCorner(3, 1);

    compute_residual_kernel<<<grid, block>>>(ref_image, ref_vmap, src_image, KR, Kt, out_image);
}

} // namespace fusion
